#include "hip/hip_runtime.h"
//
// Contains the autotuning logic and some utility functions.
// Note that all CUDA kernels have been moved to other .cu files
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=124
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <map>
#include <algorithm>

#include <hip/hip_runtime.h>

#include "salsa_kernel.h"

#include "titan_kernel.h"
#include "spinlock_kernel.h"
#include "fermi_kernel.h"
#include "legacy_kernel.h"
#include "test_kernel.h"
#include "kepler_kernel.h"

#include "miner.h"

// require CUDA 5.5 driver API
#define DMAJ 5
#define DMIN 5

// some globals containing pointers to device memory (for chunked allocation)
// [8] indexes up to 8 threads (0...7)
int       MAXWARPS[8];
uint32_t* h_V[8][1024];
uint32_t  h_V_extra[8][1024];

extern "C" int cuda_num_devices()
{
    int version;
    int err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
        exit(1);
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < DMAJ || (maj == DMAJ && min < DMIN))
    {
        applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", DMAJ, DMIN);
        exit(1);
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }
    return GPU_N;
}

bool validate_config(char *config, int &b, int &w, KernelInterface **kernel = NULL, hipDeviceProp_t *props = NULL)
{
    bool success = false;
    char kernelid = ' ';
    if (config != NULL)
    {
        if (config[0] == 'T' || config[0] == 'S' || config[0] == 'K' || config[0] == 'F' || config[0] == 'L' ||
            config[0] == 'X') {
            kernelid = config[0];
            config++;
        }

        if (config[0] >= '0' && config[0] <= '9')
            if (sscanf(config, "%dx%d", &b, &w) == 2)
                success = true;

        if (success && kernel != NULL)
        {
            switch (kernelid)
            {
                case 'T': *kernel = new TitanKernel(); break;
                case 'S': *kernel = new SpinlockKernel(); break;
                case 'K': *kernel = new KeplerKernel(); break;
                case 'F': *kernel = new FermiKernel(); break;
                case 'L': *kernel = new LegacyKernel(); break;
                case 'X': *kernel = new TestKernel(); break;
                case ' ': // choose based on device architecture
                     if (props->major == 3 && props->minor == 5)
                    *kernel = new TitanKernel();
                else if (props->major == 3 && props->minor == 0)
                    *kernel = new KeplerKernel();
                else if (props->major == 2)
                    *kernel = new FermiKernel();
                else if (props->major == 1)
                    *kernel = new LegacyKernel();
                break;
            }
        }
    }
    return success;
}

std::map<int, int> context_blocks;
std::map<int, int> context_wpb;
std::map<int, bool> context_concurrent;
std::map<int, KernelInterface *> context_kernel;
std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_X[2];
std::map<int, uint32_t *> context_H[2];
std::map<int, hipEvent_t> context_serialize[2];

// for SHA256 hashing on GPU
std::map<int, uint32_t *> context_tstate[2];
std::map<int, uint32_t *> context_ostate[2];
std::map<int, uint32_t *> context_hash[2];

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &wpb);

extern "C" void cuda_shutdown(int thr_id)
{
    checkCudaErrors(hipStreamSynchronize(context_streams[0][thr_id]));
    checkCudaErrors(hipStreamSynchronize(context_streams[1][thr_id]));
    hipDeviceReset();
}

extern "C" int cuda_throughput(int thr_id)
{
    int GRID_BLOCKS, WARPS_PER_BLOCK;
    if (context_blocks.find(thr_id) == context_blocks.end())
    {
#if 0
        hipCtx_t ctx;
        hipCtxCreate( &ctx, hipDeviceScheduleYield, device_map[thr_id] );
        hipCtxSetCurrent(ctx);
#else
        hipSetDeviceFlags(hipDeviceScheduleYield);
        hipSetDevice(device_map[thr_id]);
        hipFree(0);
#endif

        KernelInterface *kernel;
        bool concurrent; GRID_BLOCKS = find_optimal_blockcount(thr_id, kernel, concurrent, WARPS_PER_BLOCK);
        unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;
        unsigned int state_size = WU_PER_LAUNCH * sizeof(uint32_t) * 8;

        // allocate device memory for scrypt_core inputs and outputs
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[1][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[1][thr_id] = tmp;

        // allocate pinned host memory for scrypt hashes
        checkCudaErrors(hipHostAlloc((void **) &tmp, state_size, hipHostMallocDefault)); context_H[0][thr_id] = tmp;
        checkCudaErrors(hipHostAlloc((void **) &tmp, state_size, hipHostMallocDefault)); context_H[1][thr_id] = tmp;

        if (parallel < 2)
        {
            // allocate pinned host memory for scrypt_core input/output
            checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[0][thr_id] = tmp;
            checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[1][thr_id] = tmp;
        }
        else
        {
            // allocate tstate, ostate, scrypt hash device memory
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_tstate[0][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_tstate[1][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_ostate[0][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_ostate[1][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[0][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[1][thr_id] = tmp;
        }

        // create two CUDA streams
        hipStream_t tmp2;
        checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[0][thr_id] = tmp2;
        checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[1][thr_id] = tmp2;

        // events used to serialize the kernel launches (we don't want any overlapping of kernels)
        hipEvent_t tmp4;
        checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[0][thr_id] = tmp4;
        checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[1][thr_id] = tmp4;
        hipEventRecord(context_serialize[1][thr_id]);

        context_kernel[thr_id] = kernel;
        context_concurrent[thr_id] = concurrent;
        context_blocks[thr_id] = GRID_BLOCKS;
        context_wpb[thr_id] = WARPS_PER_BLOCK;
    }

    GRID_BLOCKS = context_blocks[thr_id];
    WARPS_PER_BLOCK = context_wpb[thr_id];
    return WU_PER_LAUNCH;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10, 8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11, 8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12, 8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13, 8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
//    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
    return nGpuArchCoresPerSM[7].Cores;
}

#ifdef WIN32
#include <windows.h>
static int console_width()
{
    CONSOLE_SCREEN_BUFFER_INFO csbi;
    GetConsoleScreenBufferInfo(GetStdHandle(STD_OUTPUT_HANDLE), &csbi);
    return csbi.srWindow.Right - csbi.srWindow.Left + 1;
}
#else
int console_width()
{
    return 999;
}
#endif

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &WARPS_PER_BLOCK)
{
    int cw = console_width();
    int optimal_blocks = 0;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_map[thr_id]);
    concurrent = (props.concurrentKernels > 0);

    device_name[thr_id] = strdup(props.name);
    applog(LOG_INFO, "GPU #%d: %s with compute capability %d.%d", device_map[thr_id], props.name, props.major, props.minor);

    WARPS_PER_BLOCK = -1;

    // if not specified, use interactive mode for devices that have the watchdog timer enabled
    if (device_interactive[thr_id] == -1)
        device_interactive[thr_id] = props.kernelExecTimeoutEnabled;

    // turn off texture cache if not otherwise specified
    if (device_texturecache[thr_id] == -1)
        device_texturecache[thr_id] = 0;

    // if not otherwise specified or required, turn single memory allocations off as they reduce
    // the amount of memory that we can allocate on Windows Vista, 7 and 8 (WDDM driver model issue)
    if (device_singlememory[thr_id] == -1) device_singlememory[thr_id] = 0;

    // figure out which kernel implementation to use
    if (!validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK, &kernel, &props)) {
             if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'T') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && (props.major == 3 && props.minor == 5)))
            kernel = new TitanKernel();
        else if  (device_config[thr_id] != NULL && device_config[thr_id][0] == 'S')
            kernel = new SpinlockKernel();
        else if  ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'K') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && (props.major == 3 && props.minor == 0)))
            kernel = new KeplerKernel();
        else if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'F') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && props.major == 2))
            kernel = new FermiKernel();
        else if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'L') ||
                 ((device_config[thr_id] == NULL || !strcasecmp(device_config[thr_id], "auto")) && props.major == 1))
            kernel = new LegacyKernel();
        else if ((device_config[thr_id] != NULL && device_config[thr_id][0] == 'X'))
            kernel = new TestKernel();
    }

    if (kernel->get_major_version() > props.major || kernel->get_major_version() == props.major && kernel->get_minor_version() > props.minor)
    {
        applog(LOG_ERR, "GPU #%d: the '%c' kernel requires %d.%d capability!", device_map[thr_id], kernel->get_identifier(), kernel->get_major_version(), kernel->get_minor_version());
    }

    // set whatever cache configuration and shared memory bank mode the kernel prefers
    hipDeviceSetCacheConfig(kernel->cache_config());
    hipDeviceSetSharedMemConfig(kernel->shared_mem_config());

    // some kernels (e.g. Titan) do not support the texture cache
    if (kernel->no_textures() && device_texturecache[thr_id]) {
        applog(LOG_WARNING, "GPU #%d: the '%c' kernel ignores the texture cache argument", device_map[thr_id], kernel->get_identifier());
        device_texturecache[thr_id] = 0;
    }

    // Texture caching only works with single memory allocation
    if (device_texturecache[thr_id]) device_singlememory[thr_id] = 1;

    if (kernel->single_memory() && !device_singlememory[thr_id]) {
        applog(LOG_WARNING, "GPU #%d: the '%c' kernel requires single memory allocation", device_map[thr_id], kernel->get_identifier());
        device_singlememory[thr_id] = 1;
    }

    applog(LOG_INFO, "GPU #%d: interactive: %d, tex-cache: %d%c, single-alloc: %d", device_map[thr_id],
           (device_interactive[thr_id]  != 0) ? 1 : 0,
           (device_texturecache[thr_id] != 0) ? device_texturecache[thr_id] : 0, (device_texturecache[thr_id] != 0) ? 'D' : ' ',
           (device_singlememory[thr_id] != 0) ? 1 : 0 );

    // compute highest MAXWARPS numbers for kernels allowing hipBindTexture to succeed
    int MW_1D_4 = 134217728 / (SCRATCH * WU_PER_WARP / 4); // for uint4_t textures
    int MW_1D_2 = 134217728 / (SCRATCH * WU_PER_WARP / 2); // for uint2_t textures
    int MW_1D = kernel->get_texel_width() == 2 ? MW_1D_2 : MW_1D_4;

    uint32_t *d_V = NULL;
    if (device_singlememory[thr_id])
    {
        // if no launch config was specified, we simply
        // allocate the single largest memory chunk on the device that we can get
        if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK)) {
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
        }
        else {
            // compute no. of warps to allocate the largest number producing a single memory block below 4GB
            for (int warp = 0x7FFFFFFF / (SCRATCH * WU_PER_WARP * sizeof(uint32_t)); warp >= 1; --warp) {
                hipGetLastError(); // clear the error state
                checkCudaErrors(hipMalloc((void **)&d_V, SCRATCH * WU_PER_WARP * warp * sizeof(uint32_t)));
                if (hipGetLastError() == hipSuccess) {
                    checkCudaErrors(hipFree(d_V)); d_V = NULL;
                    MAXWARPS[thr_id] = 90*warp/100; // Windows needs some breathing room to operate safely
                                                    // in particular when binding large 1D or 2D textures
                    break;
                }
            }
        }

        // now allocate a buffer for determined MAXWARPS setting
        hipGetLastError(); // clear the error state
        checkCudaErrors(hipMalloc((void **)&d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t)));
        if (hipGetLastError() == hipSuccess) {
            for (int i=0; i < MAXWARPS[thr_id]; ++i)
                h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

            if (device_texturecache[thr_id] == 1)
            {
                if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
                {
                    if ( optimal_blocks * WARPS_PER_BLOCK > MW_1D )
                        applog(LOG_ERR, "GPU #%d: Given launch config '%s' exceeds limits for 1D cache.", device_map[thr_id], device_config[thr_id]);
                }
                // bind linear memory to a 1D texture reference
                if (kernel->get_texel_width() == 2)
                    kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * std::min(MAXWARPS[thr_id],MW_1D_2) * sizeof(uint32_t));
                else
                    kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * std::min(MAXWARPS[thr_id],MW_1D_4) * sizeof(uint32_t));
            }
            else if (device_texturecache[thr_id] == 2)
            {
                // bind pitch linear memory to a 2D texture reference
                if (kernel->get_texel_width() == 2)
                    kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                else
                    kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
            }
        }
    }
    else
    {
        if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
        else
            MAXWARPS[thr_id] = 1024;

        // chunked memory allocation up to device limits
        int warp;
        for (warp = 0; warp < MAXWARPS[thr_id]; ++warp) {
            // work around partition camping problems by adding an offset
            h_V_extra[thr_id][warp] = (props.major == 1) ? (16 * (rand()%(16384/16))) : 0;
            hipGetLastError(); // clear the error state
            checkCudaErrors(hipMalloc((void **) &h_V[thr_id][warp], (SCRATCH * WU_PER_WARP + h_V_extra[thr_id][warp])*sizeof(uint32_t)));
            if (hipGetLastError() == hipSuccess) h_V[thr_id][warp] += h_V_extra[thr_id][warp];
            else {
                h_V_extra[thr_id][warp] = 0;
                // back off by two allocations to have some breathing room
                for (int i=0; warp > 0 && i < 2; ++i) {
                    warp--;
                    checkCudaErrors(hipFree(h_V[thr_id][warp]-h_V_extra[thr_id][warp]));
                    h_V[thr_id][warp] = NULL; h_V_extra[thr_id][warp] = 0;
                }
                break;
            }
        }
        MAXWARPS[thr_id] = warp;
    }
    kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);

    if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
    {
        if (optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
            applog(LOG_ERR, "GPU #%d: Given launch config '%s' requires too much memory.", device_map[thr_id], device_config[thr_id]);
    }
    else
    {
        if (device_config[thr_id] != NULL && strcasecmp("auto", device_config[thr_id]))
            applog(LOG_WARNING, "GPU #%d: Given launch config '%s' does not validate.", device_map[thr_id], device_config[thr_id]);

        if (autotune)
        {
            applog(LOG_INFO, "GPU #%d: Performing auto-tuning (Patience...)", device_map[thr_id]);

            // allocate device memory
            unsigned int mem_size = MAXWARPS[thr_id] * WU_PER_WARP * sizeof(uint32_t) * 32;
            uint32_t *d_idata;
            checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
            uint32_t *d_odata;
            checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

            // pre-initialize some device memory
            uint32_t *h_idata = (uint32_t*)malloc(mem_size);
            for (unsigned int i=0; i < mem_size/sizeof(uint32_t); ++i) h_idata[i] = i*2654435761UL; // knuth's method
            checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
            free(h_idata);

            double best_khash_sec = 0.0;
            int best_wpb = 0;

            // auto-tuning loop
            {
                // compute highest MAXWARPS number that we can support based on texture cache mode
                int MW = (device_texturecache[thr_id] == 1) ? std::min(MAXWARPS[thr_id],MW_1D) : MAXWARPS[thr_id];

                applog(LOG_INFO, "GPU #%d: maximum warps: %d", device_map[thr_id], MW);

                for (int GRID_BLOCKS = 1; !abort_flag && GRID_BLOCKS <= MW; ++GRID_BLOCKS)
                {
                    double kHash[24+1] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
                    for (WARPS_PER_BLOCK = 1; !abort_flag && WARPS_PER_BLOCK <= kernel->max_warps_per_block(); ++WARPS_PER_BLOCK)
                    {
                        double khash_sec = 0;
                        if (GRID_BLOCKS * WARPS_PER_BLOCK <= MW)
                        {
                            // setup execution parameters
                            dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
                            dim3  threads(WU_PER_BLOCK, 1, 1);

                            struct timeval tv_start, tv_end;
                            double tdelta = 0;

                            hipDeviceSynchronize();
                            gettimeofday(&tv_start, NULL);
                            int repeat = 0;
                            bool r = false;
                            while (repeat < 3)  // average up to 3 measurements for better exactness
                            {
                                r=kernel->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id, NULL, d_idata, d_odata, device_interactive[thr_id], true, device_texturecache[thr_id]);
                                hipDeviceSynchronize();
                                if (!r || hipPeekAtLastError() != hipSuccess) break;
                                ++repeat;
                                gettimeofday(&tv_end, NULL);
                                // bail out if 50ms taken (to speed up autotuning...)
                                if ((1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec)) > 0.05) break;
                            }
                            if (hipGetLastError() != hipSuccess || !r) continue;

                            tdelta = (1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec)) / repeat;

                            if (device_interactive[thr_id] && GRID_BLOCKS > 2*props.multiProcessorCount && tdelta > 1.0/30)
                                if (WARPS_PER_BLOCK == 1) goto skip; else goto skip2;

                            khash_sec = WU_PER_LAUNCH / (tdelta * 1e3);
                            kHash[WARPS_PER_BLOCK] = khash_sec;
                            if (khash_sec > best_khash_sec) {
                                optimal_blocks = GRID_BLOCKS;
                                best_khash_sec = khash_sec;
                                best_wpb = WARPS_PER_BLOCK;
                            }
                        }
                    }
skip2:              ;
                    if (opt_debug) {
                        if (GRID_BLOCKS == 1) {
                            char line[256] = "    ";
                            for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
                                char tmp[16]; sprintf(tmp, "   x%-2d", i);
                                strcat(line, tmp);
                                if (cw == 80 && (i == 8 || i == 16)) strcat(line, "\n                          ");
                            }
                            applog(LOG_DEBUG, line);
                        }
                        char line[256]; sprintf(line, "%3d:", GRID_BLOCKS);
                        for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
                            char tmp[16];
                            if (kHash[i]>0)
                                sprintf(tmp, "%5.1f%c", kHash[i], (i<kernel->max_warps_per_block())?'|':' ');
                            else
                                sprintf(tmp, "     %c", (i<kernel->max_warps_per_block())?'|':' ');
                            strcat(line, tmp);
                            if (cw == 80 && (i == 8 || i == 16)) strcat(line, "\n                          ");
                        }
                        strcat(line, "kH/s");
                        applog(LOG_DEBUG, line);
                    }
                }
skip:           ;
            }

            checkCudaErrors(hipFree(d_odata));
            checkCudaErrors(hipFree(d_idata));

            WARPS_PER_BLOCK = best_wpb;
            applog(LOG_INFO, "GPU #%d: %7.2f khash/s with configuration %c%dx%d", device_map[thr_id], best_khash_sec, kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);
        }
        else
        {
            // Heuristics for finding a good kernel launch configuration

            // base the initial block estimate on the number of multiprocessors
            int device_cores = props.multiProcessorCount * _ConvertSMVer2Cores(props.major, props.minor);

            // defaults, in case nothing else is chosen below
            optimal_blocks = 4 * device_cores / WU_PER_WARP;
            WARPS_PER_BLOCK = 2;

            // Based on compute capability, pick a known good block x warp configuration.
            if (props.major == 3)
            {
                if (props.minor == 0) // GK104, GK106, GK107
                {
                    if (MAXWARPS[thr_id] > (int)(optimal_blocks * 1.7261905) * 2)
                    {
                        // this results in 290x2 configuration on GTX 660Ti (3GB)
                        // but it requires 3GB memory on the card!
                        optimal_blocks = (int)(optimal_blocks * 1.7261905);
                        WARPS_PER_BLOCK = 2;
                    }
                    else
                    {
                        // this results in 148x2 configuration on GTX 660Ti (2GB)
                        optimal_blocks = (int)(optimal_blocks * 0.8809524);
                        WARPS_PER_BLOCK = 2;
                    }
                }
                else if (props.minor == 5) // GK110 (Tesla K20X, K20, GeForce GTX TITAN)
                {
                    // TODO: what to do with Titan and Tesla K20(X)?
                    // for now, do the same as for GTX 660Ti (2GB)
                    optimal_blocks = (int)(optimal_blocks * 0.8809524);
                    WARPS_PER_BLOCK = 2;
                }
            }
            // 1st generation Fermi (compute 2.0) GF100, GF110
            else if (props.major == 2 && props.minor == 0)
            {
                // this results in a 60x4 configuration on GTX 570
                optimal_blocks = 4 * device_cores / WU_PER_WARP;
                WARPS_PER_BLOCK = 4;
            }
            // 2nd generation Fermi (compute 2.1) GF104,106,108,114,116
            else if (props.major == 2 && props.minor == 1)
            {
                // this results in a 56x2 configuration on GTX 460
                optimal_blocks = props.multiProcessorCount * 8;
                WARPS_PER_BLOCK = 2;
            }
            // G80, G92, GT2xx
            else if (props.major == 1)
            {
                if (props.minor == 0)  // G80
                {
                    // TODO: anyone knowing good settings for G80?
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 4;
                }
                else if (props.minor == 1)  // G92
                {
                    // e.g. my 9600M works best at 4x4
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 4;
                }
                else if (props.minor == 2)  // GT218, GT216, GT215
                {
                    // TODO: anyone knowing good settings for Compute 1.2?
                    // for now I assume performance is identical to compute 1.3
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 3;
                }
                if (props.minor == 3)  // GT200
                {
                    // my GTX 260 works best at S27x3
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 3;
                }
            }

            // in case we run out of memory with the automatically chosen configuration,
            // first back off with WARPS_PER_BLOCK, then reduce optimal_blocks.
            if (WARPS_PER_BLOCK==3 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
                WARPS_PER_BLOCK = 2;
            while (optimal_blocks > 0 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
                optimal_blocks--;
        }
    }

    applog(LOG_INFO, "GPU #%d: using launch configuration %c%dx%d", device_map[thr_id], kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);

    if (device_singlememory[thr_id])
    {
        if (MAXWARPS[thr_id] != optimal_blocks * WARPS_PER_BLOCK)
        {
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
            if (device_texturecache[thr_id] == 1)
                kernel->unbindtexture_1D();
            else if (device_texturecache[thr_id] == 2)
                kernel->unbindtexture_2D();
            checkCudaErrors(hipFree(d_V)); d_V = NULL;

            hipGetLastError(); // clear the error state
            checkCudaErrors(hipMalloc((void **)&d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t)));
            if (hipGetLastError() == hipSuccess) {
                for (int i=0; i < MAXWARPS[thr_id]; ++i)
                    h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

                if (device_texturecache[thr_id] == 1)
                {
                    // bind linear memory to a 1D texture reference
                    if (kernel->get_texel_width() == 2)
                        kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
                    else
                        kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
                }
                else if (device_texturecache[thr_id] == 2)
                {
                    // bind pitch linear memory to a 2D texture reference
                    if (kernel->get_texel_width() == 2)
                        kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                    else
                        kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                }

                // update pointers to scratch buffer in constant memory after reallocation
                kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);
            }
            else
            {
                applog(LOG_ERR, "GPU #%d: Unable to allocate enough memory for launch config '%s'.", device_map[thr_id], device_config[thr_id]);
            }
        }
    }
    else
    {
        // back off unnecessary memory allocations to have some breathing room
        while (MAXWARPS[thr_id] > 0 && MAXWARPS[thr_id] > optimal_blocks * WARPS_PER_BLOCK) {
            (MAXWARPS[thr_id])--;
            checkCudaErrors(hipFree(h_V[thr_id][MAXWARPS[thr_id]]-h_V_extra[thr_id][MAXWARPS[thr_id]]));
            h_V[thr_id][MAXWARPS[thr_id]] = NULL; h_V_extra[thr_id][MAXWARPS[thr_id]] = 0;
        }
    }

    return optimal_blocks;
}

hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id)
{
    hipError_t result = hipSuccess;
    static double tsum[3][8] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    double tsync = 0.0;
    double tsleep = 0.95 * tsum[situation][thr_id];
    if (hipStreamQuery(stream) == hipErrorNotReady)
    {
#ifdef WIN32
        Sleep((DWORD)(1000*tsleep));
#else
        usleep((useconds_t)(1e6*tsleep));
#endif
        struct timeval tv_start, tv_end;
        gettimeofday(&tv_start, NULL);
        checkCudaErrors(result = hipStreamSynchronize(stream));
        gettimeofday(&tv_end, NULL);
        tsync = 1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec);
    }
    if (tsync >= 0) tsum[situation][thr_id] = 0.95 * tsum[situation][thr_id] + 0.05 * (tsleep+tsync);

    return result;
}

extern "C" void cuda_scrypt_HtoD(int thr_id, uint32_t *X, int stream)
{
    int GRID_BLOCKS = context_blocks[thr_id];
    int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

    // copy host memory to device
    checkCudaErrors(hipMemcpyAsync(context_idata[stream][thr_id], X, mem_size,
                               hipMemcpyHostToDevice, context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_serialize(int thr_id, int stream)
{
    // if the device can concurrently execute multiple kernels, then we must
    // wait for the serialization event recorded by the other stream
    if (context_concurrent[thr_id] || device_interactive[thr_id])
        checkCudaErrors(hipStreamWaitEvent(context_streams[stream][thr_id], context_serialize[(stream+1)&1][thr_id], 0));
}

extern "C" void cuda_scrypt_done(int thr_id, int stream)
{
    // record the serialization event in the current stream
    checkCudaErrors(hipEventRecord(context_serialize[stream][thr_id], context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_flush(int thr_id, int stream)
{
    // flush the work queue (required for WDDM drivers)
    checkCudaErrors(hipStreamQuery(context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_core(int thr_id, int stream)
{
    int GRID_BLOCKS = context_blocks[thr_id];
    int WARPS_PER_BLOCK = context_wpb[thr_id];

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(WU_PER_BLOCK, 1, 1);

    if (device_interactive[thr_id]) {
//        checkCudaErrors(MyStreamSynchronize(context_streams[stream][thr_id], 2, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    context_kernel[thr_id]->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id, context_streams[stream][thr_id], context_idata[stream][thr_id], context_odata[stream][thr_id], device_interactive[thr_id], false, device_texturecache[thr_id]);
}

extern "C" void cuda_scrypt_DtoH(int thr_id, uint32_t *X, int stream)
{
    int GRID_BLOCKS = context_blocks[thr_id];
    int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

    // copy result from device to host (asynchronously)
    checkCudaErrors(hipMemcpyAsync(X, context_odata[stream][thr_id], mem_size,
                               hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
}

extern "C" void cuda_scrypt_sync(int thr_id, int stream)
{
    MyStreamSynchronize(context_streams[stream][thr_id], 0, thr_id);
}

extern "C" uint32_t* cuda_transferbuffer(int thr_id, int stream)
{
    return context_X[stream][thr_id];
}

extern "C" uint32_t* cuda_hashbuffer(int thr_id, int stream)
{
    return context_H[stream][thr_id];
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on the CPU
//! @param idata      input data as provided to device
//! @param reference  reference data, computed but preallocated
//! @param V          scrypt scratchpad
////////////////////////////////////////////////////////////////////////////////
static void xor_salsa8(uint32_t * const B, const uint32_t * const C);

extern "C" void
computeGold(uint32_t *idata, uint32_t *reference, uint32_t *V)
{
	uint32_t X[32];
	int i,j,k;

	for (k = 0; k < 32; k++)
		X[k] = idata[k];
	
	for (i = 0; i < 1024; i++) {
		memcpy(&V[i * 32], X, 128);
		xor_salsa8(&X[0], &X[16]);
		xor_salsa8(&X[16], &X[0]);
	}
	for (i = 0; i < 1024; i++) {
		j = 32 * (X[16] & 1023);
		for (k = 0; k < 32; k++)
			X[k] ^= V[j + k];
		xor_salsa8(&X[0], &X[16]);
		xor_salsa8(&X[16], &X[0]);
	}
	for (k = 0; k < 32; k++)
		reference[k] = X[k];
}

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

static void xor_salsa8(uint32_t * const B, const uint32_t * const C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);

    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);

    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

//
//  =============== SHA256 part ======================
//

static const uint32_t host_sha256_h[8] = {
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
	0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};

static const uint32_t host_sha256_k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

/* Elementary functions used by SHA256 */
#define Ch(x, y, z)     ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define ROTR(x, n)      ((x >> n) | (x << (32 - n)))
#define S0(x)           (ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22))
#define S1(x)           (ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25))
#define s0(x)           (ROTR(x, 7) ^ ROTR(x, 18) ^ (x >> 3))
#define s1(x)           (ROTR(x, 17) ^ ROTR(x, 19) ^ (x >> 10))

/* SHA256 round function */
#define RND(a, b, c, d, e, f, g, h, k) \
	do { \
		t0 = h + S1(e) + Ch(e, f, g) + k; \
		t1 = S0(a) + Maj(a, b, c); \
		d += t0; \
		h  = t0 + t1; \
	} while (0)

/* Adjusted round function for rotating state */
#define RNDr(S, W, i) \
	RND(S[(64 - i) % 8], S[(65 - i) % 8], \
	    S[(66 - i) % 8], S[(67 - i) % 8], \
	    S[(68 - i) % 8], S[(69 - i) % 8], \
	    S[(70 - i) % 8], S[(71 - i) % 8], \
	    W[i] + sha256_k[i])

static const uint32_t host_keypad[12] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000280
};

static const uint32_t host_innerpad[11] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x000004a0
};

static const uint32_t host_outerpad[8] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0x00000300
};

static const uint32_t host_finalblk[16] = {
	0x00000001, 0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000620
};

//
// CUDA code
//

__constant__ uint32_t sha256_h[8];
__constant__ uint32_t sha256_k[64];
__constant__ uint32_t keypad[12];
__constant__ uint32_t innerpad[11];
__constant__ uint32_t outerpad[8];
__constant__ uint32_t finalblk[16];
__constant__ uint32_t pdata[20];
__constant__ uint32_t midstate[8];

__device__ void mycpy16(uint32_t *d, const uint32_t *s) {
#pragma unroll 4
    for (int k=0; k < 4; k++) d[k] = s[k];
}

__device__ void mycpy32(uint32_t *d, const uint32_t *s) {
#pragma unroll 8
    for (int k=0; k < 8; k++) d[k] = s[k];
}

__device__ void mycpy44(uint32_t *d, const uint32_t *s) {
#pragma unroll 11
    for (int k=0; k < 11; k++) d[k] = s[k];
}

__device__ void mycpy48(uint32_t *d, const uint32_t *s) {
#pragma unroll 12
    for (int k=0; k < 12; k++) d[k] = s[k];
}

__device__ void mycpy64(uint32_t *d, const uint32_t *s) {
#pragma unroll 16
    for (int k=0; k < 16; k++) d[k] = s[k];
}

__device__ void mycpy76(uint32_t *d, const uint32_t *s) {
#pragma unroll 19
    for (int k=0; k < 19; k++) d[k] = s[k];
}

__device__ void mycpy128(uint32_t *d, const uint32_t *s) {
#pragma unroll 32
    for (int k=0; k < 32; k++) d[k] = s[k];
}

__device__ void cuda_sha256_init(uint32_t *state)
{
	mycpy32(state, sha256_h);
}

__device__ uint32_t cuda_swab32(uint32_t x)
{
    return ((((x) << 24) & 0xff000000u) | (((x) << 8) & 0x00ff0000u) \
          | (((x) >> 8) & 0x0000ff00u) | (((x) >> 24) & 0x000000ffu));
}

/*
 * SHA256 block compression function.  The 256-bit state is transformed via
 * the 512-bit input block to produce a new state.
 */
__device__ void cuda_sha256_transform(uint32_t *state, const uint32_t *block, int swap)
{
	uint32_t W[64];
	uint32_t S[8];
	uint32_t t0, t1;
	int i;

	/* 1. Prepare message schedule W. */
	if (swap) {
#pragma unroll 16
		for (i = 0; i < 16; i++)
			W[i] = cuda_swab32(block[i]);
	} else
		mycpy64(W, block);
#pragma unroll 24
	for (i = 16; i < 64; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15];
	}

	/* 2. Initialize working variables. */
	mycpy32(S, state);

	/* 3. Mix. */
	RNDr(S, W,  0);
	RNDr(S, W,  1);
	RNDr(S, W,  2);
	RNDr(S, W,  3);
	RNDr(S, W,  4);
	RNDr(S, W,  5);
	RNDr(S, W,  6);
	RNDr(S, W,  7);
	RNDr(S, W,  8);
	RNDr(S, W,  9);
	RNDr(S, W, 10);
	RNDr(S, W, 11);
	RNDr(S, W, 12);
	RNDr(S, W, 13);
	RNDr(S, W, 14);
	RNDr(S, W, 15);
	RNDr(S, W, 16);
	RNDr(S, W, 17);
	RNDr(S, W, 18);
	RNDr(S, W, 19);
	RNDr(S, W, 20);
	RNDr(S, W, 21);
	RNDr(S, W, 22);
	RNDr(S, W, 23);
	RNDr(S, W, 24);
	RNDr(S, W, 25);
	RNDr(S, W, 26);
	RNDr(S, W, 27);
	RNDr(S, W, 28);
	RNDr(S, W, 29);
	RNDr(S, W, 30);
	RNDr(S, W, 31);
	RNDr(S, W, 32);
	RNDr(S, W, 33);
	RNDr(S, W, 34);
	RNDr(S, W, 35);
	RNDr(S, W, 36);
	RNDr(S, W, 37);
	RNDr(S, W, 38);
	RNDr(S, W, 39);
	RNDr(S, W, 40);
	RNDr(S, W, 41);
	RNDr(S, W, 42);
	RNDr(S, W, 43);
	RNDr(S, W, 44);
	RNDr(S, W, 45);
	RNDr(S, W, 46);
	RNDr(S, W, 47);
	RNDr(S, W, 48);
	RNDr(S, W, 49);
	RNDr(S, W, 50);
	RNDr(S, W, 51);
	RNDr(S, W, 52);
	RNDr(S, W, 53);
	RNDr(S, W, 54);
	RNDr(S, W, 55);
	RNDr(S, W, 56);
	RNDr(S, W, 57);
	RNDr(S, W, 58);
	RNDr(S, W, 59);
	RNDr(S, W, 60);
	RNDr(S, W, 61);
	RNDr(S, W, 62);
	RNDr(S, W, 63);

	/* 4. Mix local working variables into global state */
#pragma unroll 8
	for (i = 0; i < 8; i++)
		state[i] += S[i];
}

//
// Original scrypt.cpp HMAC SHA256 functions
//

__device__ void cuda_HMAC_SHA256_80_init(const uint32_t *key,
	uint32_t *tstate, uint32_t *ostate)
{
	uint32_t ihash[8];
	uint32_t pad[16];
	int i;

	/* tstate is assumed to contain the midstate of key */
	mycpy16(pad, key + 16);
	mycpy48(pad + 4, keypad);
	cuda_sha256_transform(tstate, pad, 0);
	mycpy32(ihash, tstate);

	cuda_sha256_init(ostate);
#pragma unroll 8
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x5c5c5c5c;
#pragma unroll 8
	for (i=8; i < 16; i++)
		pad[i] = 0x5c5c5c5c;
	cuda_sha256_transform(ostate, pad, 0);

	cuda_sha256_init(tstate);
#pragma unroll 8
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x36363636;
#pragma unroll 8
	for (i=8; i < 16; i++)
		pad[i] = 0x36363636;
	cuda_sha256_transform(tstate, pad, 0);
}

__device__ void cuda_PBKDF2_SHA256_80_128(const uint32_t *tstate,
	const uint32_t *ostate, const uint32_t *salt, uint32_t *output)
{
	uint32_t istate[8], ostate2[8];
	uint32_t ibuf[16], obuf[16];
	int j;

	mycpy32(istate, tstate);
	cuda_sha256_transform(istate, salt, 0);
	
	mycpy16(ibuf, salt + 16);
	mycpy44(ibuf + 5, innerpad);
	mycpy32(obuf + 8, outerpad);

	mycpy32(obuf, istate);
	ibuf[4] = 1;
	cuda_sha256_transform(obuf, ibuf, 0);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf, 0);
#pragma unroll 8
	for (j = 0; j < 8; j++)
		output[0 + j] = cuda_swab32(ostate2[j]); // TODO: coalescing!

	mycpy32(obuf, istate);
	ibuf[4] = 2;
	cuda_sha256_transform(obuf, ibuf, 0);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf, 0);
#pragma unroll 8
	for (j = 0; j < 8; j++)
		output[8 + j] = cuda_swab32(ostate2[j]); // TODO: coalescing!

	mycpy32(obuf, istate);
	ibuf[4] = 3;
	cuda_sha256_transform(obuf, ibuf, 0);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf, 0);
#pragma unroll 8
	for (j = 0; j < 8; j++)
		output[16 + j] = cuda_swab32(ostate2[j]); // TODO: coalescing!

	mycpy32(obuf, istate);
	ibuf[4] = 4;
	cuda_sha256_transform(obuf, ibuf, 0);

	mycpy32(ostate2, ostate);
	cuda_sha256_transform(ostate2, obuf, 0);
#pragma unroll 8
	for (j = 0; j < 8; j++)
		output[24 + j] = cuda_swab32(ostate2[j]); // TODO: coalescing!
}

extern "C" void prepare_sha256(int thr_id, uint32_t host_pdata[20], uint32_t host_midstate[8])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        hipMemcpyToSymbol(HIP_SYMBOL(sha256_h), host_sha256_h, sizeof(host_sha256_h), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(sha256_k), host_sha256_k, sizeof(host_sha256_k), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(keypad), host_keypad, sizeof(host_keypad), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(innerpad), host_innerpad, sizeof(host_innerpad), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(outerpad), host_outerpad, sizeof(host_outerpad), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(finalblk), host_finalblk, sizeof(host_finalblk), 0, hipMemcpyHostToDevice);
        init[thr_id] = true;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(midstate), host_midstate, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

__global__ void cuda_pre_sha256(uint32_t inp[32], uint32_t tstate_ext[8], uint32_t ostate_ext[8], uint32_t nonce)
{
	nonce += (blockIdx.x * blockDim.x) + threadIdx.x; 
	inp += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	tstate_ext += 8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	ostate_ext += 8 * ((blockIdx.x * blockDim.x) + threadIdx.x);

	uint32_t ldata[20], tstate[8], ostate[8];
	mycpy76(ldata, pdata); ldata[19] = nonce;
	mycpy32(tstate, midstate);
	cuda_HMAC_SHA256_80_init(ldata, tstate, ostate);
	cuda_PBKDF2_SHA256_80_128(tstate, ostate, ldata, inp);

	// TODO: coalescing would be desired
	mycpy32(tstate_ext, tstate);
	mycpy32(ostate_ext, ostate);
}

extern "C" void pre_sha256(int thr_id, int stream, uint32_t nonce, int throughput)
{
    dim3 block(32);
    dim3 grid((throughput+31)/32);

    cuda_pre_sha256<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_idata[stream][thr_id], context_tstate[stream][thr_id], context_ostate[stream][thr_id], nonce);
}

__global__ void cuda_post_sha256(uint32_t output[8], uint32_t tstate_ext[8], uint32_t ostate_ext[8], uint32_t salt_ext[32])
{
	output += 8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	tstate_ext += 8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	ostate_ext += 8 * ((blockIdx.x * blockDim.x) + threadIdx.x);
	salt_ext += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);

	uint32_t tstate[16];
	uint32_t salt[32];
	uint32_t buf[16];
	uint32_t ostate[16];
	int i;
	
	// TODO: coalescing would be desired
	mycpy32(tstate, tstate_ext);
	mycpy32(ostate, ostate_ext);
	mycpy128(salt, salt_ext);
	
	cuda_sha256_transform(tstate, salt, 1);
	cuda_sha256_transform(tstate, salt + 16, 1);
	cuda_sha256_transform(tstate, finalblk, 0);
	mycpy32(buf, tstate);
	mycpy32(buf + 8, outerpad);

	cuda_sha256_transform(ostate, buf, 0);
#pragma unroll 8
	for (i = 0; i < 8; i++)
		output[i] = cuda_swab32(ostate[i]); // TODO: coalescing
}

extern "C" void post_sha256(int thr_id, int stream, uint32_t hash[8], int throughput)
{
    dim3 block(32);
    dim3 grid((throughput+31)/32);

    cuda_post_sha256<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_hash[stream][thr_id], context_tstate[stream][thr_id], context_ostate[stream][thr_id], context_odata[stream][thr_id]);

    unsigned int mem_size = throughput * sizeof(uint32_t) * 8;

    // copy device memory to host
    checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                    hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
}

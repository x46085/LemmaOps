#include "hip/hip_runtime.h"
//
// Kernel that runs best on Legacy (Compute 1.x) devices
//
// -full half-warp based memory coalescing
// -high consumption of shared memory
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "legacy_kernel.h"

#if WIN32
#ifdef _WIN64
#define _64BIT_ALIGN 1
#else
#define _64BIT_ALIGN 0
#endif
#else
#if __x86_64__
#define _64BIT_ALIGN 1
#else
#define _64BIT_ALIGN 0
#endif
#endif

// forward references
template <int WARPS_PER_BLOCK> __global__ void legacy_scrypt_core_kernelA(uint32_t *g_idata);
template <int WARPS_PER_BLOCK> __global__ void legacy_scrypt_core_kernelB(uint32_t *g_odata);
template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void legacy_scrypt_core_kernelB_tex(uint32_t *g_odata);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[1024];

// using texture references for the "tex" variants of the B kernels
texture<uint2, 1, hipReadModeElementType> texRef1D_2_V;
texture<uint2, 2, hipReadModeElementType> texRef2D_2_V;

LegacyKernel::LegacyKernel() : KernelInterface()
{
}

bool LegacyKernel::bindtexture_1D(uint32_t *d_V, size_t size)
{
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<uint2>();
    texRef1D_2_V.normalized = 0;
    texRef1D_2_V.filterMode = hipFilterModePoint;
    texRef1D_2_V.addressMode[0] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture(NULL, &texRef1D_2_V, d_V, &channelDesc2, size));
    return true;
}

bool LegacyKernel::bindtexture_2D(uint32_t *d_V, int width, int height, size_t pitch)
{
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<uint2>();
    texRef2D_2_V.normalized = 0;
    texRef2D_2_V.filterMode = hipFilterModePoint;
    texRef2D_2_V.addressMode[0] = hipAddressModeClamp;
    texRef2D_2_V.addressMode[1] = hipAddressModeClamp;
    checkCudaErrors(hipBindTexture2D(NULL, &texRef2D_2_V, d_V, &channelDesc2, width, height, pitch));
    return true;
}

bool LegacyKernel::unbindtexture_1D()
{
    checkCudaErrors(hipUnbindTexture(texRef1D_2_V));
    return true;
}

bool LegacyKernel::unbindtexture_2D()
{
    checkCudaErrors(hipUnbindTexture(texRef2D_2_V));
    return true;
}

void LegacyKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool LegacyKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // First phase: Sequential writes to scratchpad.

    switch (WARPS_PER_BLOCK) {
        case 1: legacy_scrypt_core_kernelA<1><<< grid, threads, 0, stream >>>(d_idata); break;
        case 2: legacy_scrypt_core_kernelA<2><<< grid, threads, 0, stream >>>(d_idata); break;
        case 3: legacy_scrypt_core_kernelA<3><<< grid, threads, 0, stream >>>(d_idata); break;
#if EXTRA_WARPS
            case 4: legacy_scrypt_core_kernelA<4><<< grid, threads, 0, stream >>>(d_idata); break;
            case 5: legacy_scrypt_core_kernelA<5><<< grid, threads, 0, stream >>>(d_idata); break;
            case 6: legacy_scrypt_core_kernelA<6><<< grid, threads, 0, stream >>>(d_idata); break;
            case 7: legacy_scrypt_core_kernelA<7><<< grid, threads, 0, stream >>>(d_idata); break;
            case 8: legacy_scrypt_core_kernelA<8><<< grid, threads, 0, stream >>>(d_idata); break;
#endif
        default: success = false; break;
    }

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    if (texture_cache)
    {
        if (texture_cache == 1)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: legacy_scrypt_core_kernelB_tex<1,1><<< grid, threads, 0, stream >>>(d_odata); break;
                case 2: legacy_scrypt_core_kernelB_tex<2,1><<< grid, threads, 0, stream >>>(d_odata); break;
                case 3: legacy_scrypt_core_kernelB_tex<3,1><<< grid, threads, 0, stream >>>(d_odata); break;
#if EXTRA_WARPS
                    case 4: legacy_scrypt_core_kernelB_tex<4,1><<< grid, threads, 0, stream >>>(d_odata); break;
                    case 5: legacy_scrypt_core_kernelB_tex<5,1><<< grid, threads, 0, stream >>>(d_odata); break;
                    case 6: legacy_scrypt_core_kernelB_tex<6,1><<< grid, threads, 0, stream >>>(d_odata); break;
                    case 7: legacy_scrypt_core_kernelB_tex<7,1><<< grid, threads, 0, stream >>>(d_odata); break;
                    case 8: legacy_scrypt_core_kernelB_tex<8,1><<< grid, threads, 0, stream >>>(d_odata); break;
#endif
                default: success = false; break;
            }
        }
        else if (texture_cache == 2)
        {
            switch (WARPS_PER_BLOCK) {
                case 1: legacy_scrypt_core_kernelB_tex<1,2><<< grid, threads, 0, stream >>>(d_odata); break;
                case 2: legacy_scrypt_core_kernelB_tex<2,2><<< grid, threads, 0, stream >>>(d_odata); break;
                case 3: legacy_scrypt_core_kernelB_tex<3,2><<< grid, threads, 0, stream >>>(d_odata); break;
#if EXTRA_WARPS
                   case 4: legacy_scrypt_core_kernelB_tex<4,2><<< grid, threads, 0, stream >>>(d_odata); break;
                   case 5: legacy_scrypt_core_kernelB_tex<5,2><<< grid, threads, 0, stream >>>(d_odata); break;
                   case 6: legacy_scrypt_core_kernelB_tex<6,2><<< grid, threads, 0, stream >>>(d_odata); break;
                   case 7: legacy_scrypt_core_kernelB_tex<7,2><<< grid, threads, 0, stream >>>(d_odata); break;
                   case 8: legacy_scrypt_core_kernelB_tex<8,2><<< grid, threads, 0, stream >>>(d_odata); break;
#endif
                default: success = false; break;
            }
        } else success = false;
    }
    else
    {
        switch (WARPS_PER_BLOCK) {
            case 1: legacy_scrypt_core_kernelB<1><<< grid, threads, 0, stream >>>(d_odata); break;
            case 2: legacy_scrypt_core_kernelB<2><<< grid, threads, 0, stream >>>(d_odata); break;
            case 3: legacy_scrypt_core_kernelB<3><<< grid, threads, 0, stream >>>(d_odata); break;
#if EXTRA_WARPS
                case 4: legacy_scrypt_core_kernelB<4><<< grid, threads, 0, stream >>>(d_odata); break;
                case 5: legacy_scrypt_core_kernelB<5><<< grid, threads, 0, stream >>>(d_odata); break;
                case 6: legacy_scrypt_core_kernelB<6><<< grid, threads, 0, stream >>>(d_odata); break;
                case 7: legacy_scrypt_core_kernelB<7><<< grid, threads, 0, stream >>>(d_odata); break;
                case 8: legacy_scrypt_core_kernelB<8><<< grid, threads, 0, stream >>>(d_odata); break;
#endif
            default: success = false; break;
        }
    }

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<7) | ((a00)>>25) );\
a1^=(((a10)<<7) | ((a10)>>25) );\
a2^=(((a20)<<7) | ((a20)>>25) );\
a3^=(((a30)<<7) | ((a30)>>25) );\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<9) | ((a00)>>23) );\
a1^=(((a10)<<9) | ((a10)>>23) );\
a2^=(((a20)<<9) | ((a20)>>23) );\
a3^=(((a30)<<9) | ((a30)>>23) );\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<13) | ((a00)>>19) );\
a1^=(((a10)<<13) | ((a10)>>19) );\
a2^=(((a20)<<13) | ((a20)>>19) );\
a3^=(((a30)<<13) | ((a30)>>19) );\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=(((a00)<<18) | ((a00)>>14) );\
a1^=(((a10)<<18) | ((a10)>>14) );\
a2^=(((a20)<<18) | ((a20)>>14) );\
a3^=(((a30)<<18) | ((a30)>>14) );\
};\

static __host__ __device__ void xor_salsa8(uint32_t * const B, const uint32_t * const C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
	ROTL7(x4,x9,xe,x3,x0+xc,x1+x5,x6+xa,xb+xf);
	ROTL9(x8,xd,x2,x7,x0+x4,x5+x9,xa+xe,x3+xf);
	ROTL13(xc,x1,x6,xb,x4+x8,x9+xd,x2+xe,x3+x7);
	ROTL18(x0,x5,xa,xf,x8+xc,x1+xd,x2+x6,x7+xb);

    /* Operate on rows. */
	ROTL7(x1,x6,xb,xc,x0+x3,x4+x5,x9+xa,xe+xf);
	ROTL9(x2,x7,x8,xd,x0+x1,x5+x6,xa+xb,xc+xf);
	ROTL13(x3,x4,x9,xe,x1+x2,x6+x7,x8+xb,xc+xd);
	ROTL18(x0,x5,xa,xf,x2+x3,x4+x7,x8+x9,xd+xe);

    /* Operate on columns. */
	ROTL7(x4,x9,xe,x3,x0+xc,x1+x5,x6+xa,xb+xf);
	ROTL9(x8,xd,x2,x7,x0+x4,x5+x9,xa+xe,x3+xf);
	ROTL13(xc,x1,x6,xb,x4+x8,x9+xd,x2+xe,x3+x7);
	ROTL18(x0,x5,xa,xf,x8+xc,x1+xd,x2+x6,x7+xb);

    /* Operate on rows. */
	ROTL7(x1,x6,xb,xc,x0+x3,x4+x5,x9+xa,xe+xf);
	ROTL9(x2,x7,x8,xd,x0+x1,x5+x6,xa+xb,xc+xf);
	ROTL13(x3,x4,x9,xe,x1+x2,x6+x7,x8+xb,xc+xd);
	ROTL18(x0,x5,xa,xf,x2+x3,x4+x7,x8+x9,xd+xe);

    /* Operate on columns. */
	ROTL7(x4,x9,xe,x3,x0+xc,x1+x5,x6+xa,xb+xf);
	ROTL9(x8,xd,x2,x7,x0+x4,x5+x9,xa+xe,x3+xf);
	ROTL13(xc,x1,x6,xb,x4+x8,x9+xd,x2+xe,x3+x7);
	ROTL18(x0,x5,xa,xf,x8+xc,x1+xd,x2+x6,x7+xb);

    /* Operate on rows. */
	ROTL7(x1,x6,xb,xc,x0+x3,x4+x5,x9+xa,xe+xf);
	ROTL9(x2,x7,x8,xd,x0+x1,x5+x6,xa+xb,xc+xf);
	ROTL13(x3,x4,x9,xe,x1+x2,x6+x7,x8+xb,xc+xd);
	ROTL18(x0,x5,xa,xf,x2+x3,x4+x7,x8+x9,xd+xe);

    /* Operate on columns. */
	ROTL7(x4,x9,xe,x3,x0+xc,x1+x5,x6+xa,xb+xf);
	ROTL9(x8,xd,x2,x7,x0+x4,x5+x9,xa+xe,x3+xf);
	ROTL13(xc,x1,x6,xb,x4+x8,x9+xd,x2+xe,x3+x7);
	ROTL18(x0,x5,xa,xf,x8+xc,x1+xd,x2+x6,x7+xb);

    /* Operate on rows. */
	ROTL7(x1,x6,xb,xc,x0+x3,x4+x5,x9+xa,xe+xf);
	ROTL9(x2,x7,x8,xd,x0+x1,x5+x6,xa+xb,xc+xf);
	ROTL13(x3,x4,x9,xe,x1+x2,x6+x7,x8+xb,xc+xd);
	ROTL18(x0,x5,xa,xf,x2+x3,x4+x7,x8+x9,xd+xe);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

static __host__ __device__ uint2& operator^=(uint2& left, const uint2& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    return left;
}

////////////////////////////////////////////////////////////////////////////////
//! Scrypt core kernel with higher shared memory use (faster on older devices)
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int WARPS_PER_BLOCK> __global__ void
legacy_scrypt_core_kernelA(uint32_t *g_idata)
{
    __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][32+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx    = threadIdx.x / warpSize;
    volatile int warpThread = threadIdx.x % warpSize;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/16;
    unsigned int Z = 2*(warpThread%16);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_idata += 32      * offset;
    uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

    uint32_t ((*XB)[32+1+_64BIT_ALIGN]) = (uint32_t (*)[32+1+_64BIT_ALIGN])&X[warpIdx][Y][Z];
    uint32_t *XX = X[warpIdx][warpThread];

    {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)XB[wu]) = *((uint2*)(&g_idata[32*(wu+Y)+Z]));

#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&V[SCRATCH*wu])) = *((uint2*)XB[wu]);

        for (int i = 1; i < 1024; i++)
        {
            xor_salsa8(&XX[0], &XX[16]);
            xor_salsa8(&XX[16], &XX[0]);

#pragma unroll 16
            for (int wu=0; wu < 32; wu+=2)
                *((uint2*)(&V[SCRATCH*wu + i*32])) = *((uint2*)XB[wu]);
        }
    }
}

template <int WARPS_PER_BLOCK> __global__ void
legacy_scrypt_core_kernelB(uint32_t *g_odata)
{
    __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][32+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx    = threadIdx.x / warpSize;
    volatile int warpThread = threadIdx.x % warpSize;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/16;
    unsigned int Z = 2*(warpThread%16);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32      * offset;
    uint32_t * V = c_V[offset / WU_PER_WARP] + SCRATCH*Y + Z;

    uint32_t ((*XB)[32+1+_64BIT_ALIGN]) = (uint32_t (*)[32+1+_64BIT_ALIGN])&X[warpIdx][Y][Z];
    uint32_t *XX = X[warpIdx][warpThread];

    {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)XB[wu]) = *((uint2*)(&V[SCRATCH*wu + 1023*32]));

        xor_salsa8(&XX[0], &XX[16]);
        xor_salsa8(&XX[16], &XX[0]);

        for (int i = 0; i < 1024; i++)
        {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)XB[wu]) ^= *((uint2*)(&V[SCRATCH*wu + 32*(X[warpIdx][wu+Y][16] & 1023)]));

            xor_salsa8(&XX[0], &XX[16]);
            xor_salsa8(&XX[16], &XX[0]);
        }

#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)XB[wu]);
    }
}

template <int WARPS_PER_BLOCK, int TEX_DIM> __global__ void
legacy_scrypt_core_kernelB_tex(uint32_t *g_odata)
{
    __shared__ uint32_t X[WARPS_PER_BLOCK][WU_PER_WARP][32+1+_64BIT_ALIGN]; // +1 to resolve bank conflicts

    volatile int warpIdx    = threadIdx.x / warpSize;
    volatile int warpThread = threadIdx.x % warpSize;

    // variables supporting the large memory transaction magic
    unsigned int Y = warpThread/16;
    unsigned int Z = 2*(warpThread%16);

    // add block specific offsets
    int offset = blockIdx.x * WU_PER_BLOCK + warpIdx * WU_PER_WARP;
    g_odata += 32      * offset;

    uint32_t ((*XB)[32+1+_64BIT_ALIGN]) = (uint32_t (*)[32+1+_64BIT_ALIGN])&X[warpIdx][Y][Z];
    uint32_t *XX = X[warpIdx][warpThread];

    {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)XB[wu]) = ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_2_V, (SCRATCH*(offset+wu+Y) + 1023*32 + Z)/2) :
                        tex2D(texRef2D_2_V, 0.5f + (32*1023 + Z)/2, 0.5f + (offset+wu+Y)));

        xor_salsa8(&XX[0], &XX[16]);
        xor_salsa8(&XX[16], &XX[0]);

        for (int i = 0; i < 1024; i++)
        {
#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)XB[wu]) ^= ((TEX_DIM == 1) ?
                        tex1Dfetch(texRef1D_2_V, (SCRATCH*(offset+wu+Y) + 32*(X[warpIdx][wu+Y][16] & 1023) + Z)/2) :
                        tex2D(texRef2D_2_V, 0.5f + (32*(X[warpIdx][wu+Y][16] & 1023) + Z)/2, 0.5f + (offset+wu+Y)));

            xor_salsa8(&XX[0], &XX[16]);
            xor_salsa8(&XX[16], &XX[0]);
        }

#pragma unroll 16
        for (int wu=0; wu < 32; wu+=2)
            *((uint2*)(&g_odata[32*(wu+Y)+Z])) = *((uint2*)XB[wu]);
    }
}


#include "hip/hip_runtime.h"
/* Copyright (C) 2013 David G. Andersen. All rights reserved.
 * with modifications by Christian Buchner
 *
 * Use of this code is covered under the Apache 2.0 license, which
 * can be found in the file "LICENSE"
 */

// TODO: support for chunked memory allocation
//       support for 1D and 2D texture cache on Compute 3.0 devices
//       attempt V.Volkov style ILP (factor 4)

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>

#include "kepler_kernel.h"

static const int THREADS_PER_SCRYPT_BLOCK = 4;
static const int SCRYPT_SCRATCH_PER_BLOCK = (32*1024);

#if __CUDA_ARCH__ < 350 
    // Kepler (Compute 3.0)
    #define __ldg(x) (*(x))
    #define XOR_ROTATE_ADD(dst, s1, s2, amt) { uint32_t tmp = x[s1]+x[s2]; x[dst] ^= ((tmp<<amt)|(tmp>>(32-amt))); }
#else
    // Kepler (Compute 3.5)
    #define ROTL(a, b) __funnelshift_l( a, a, b );
    #define XOR_ROTATE_ADD(dst, s1, s2, amt) x[dst] ^= ROTL(x[s1]+x[s2], amt);
#endif

/* write_keys writes the 8 keys being processed by a warp to the global
 * scratchpad. To effectively use memory bandwidth, it performs the writes
 * (and reads, for read_keys) 128 bytes at a time per memory location
 * by __shfl'ing the 4 entries in bx to the threads in the next-up
 * thread group. It then has eight threads together perform uint4
 * (128 bit) writes to the destination region. This seems to make
 * quite effective use of memory bandwidth. An approach that spread
 * uint32s across more threads was slower because of the increased
 * computation it required.
 *
 * "start" is the loop iteration producing the write - the offset within
 * the block's memory.
 *
 * Internally, this algorithm first __shfl's the 4 bx entries to
 * the next up thread group, and then uses a conditional move to
 * ensure that odd-numbered thread groups exchange the b/bx ordering
 * so that the right parts are written together.
 *
 * Thanks to Babu for helping design the 128-bit-per-write version.
 *
 * _direct lets the caller specify the absolute start location instead of
 * the relative start location, as an attempt to reduce some recomputation.
 */

__device__ __forceinline__
void write_keys_direct(const uint32_t b[4], const uint32_t bx[4], uint32_t *scratch, uint32_t start) {

  uint4 t, t2;
  t.x = b[0]; t.y = b[1]; t.z = b[2]; t.w = b[3];

  int target_thread = (threadIdx.x + 4)%32;
  t2.x = __shfl((int)bx[0], target_thread);
  t2.y = __shfl((int)bx[1], target_thread);
  t2.z = __shfl((int)bx[2], target_thread);
  t2.w = __shfl((int)bx[3], target_thread);

  int t2_start = __shfl((int)start, target_thread) + 4;

  bool c = (threadIdx.x & 0x4);

  int loc = c ? t2_start : start;
  *((uint4 *)(&scratch[loc])) = (c ? t2 : t);
  loc = c ? start : t2_start;
  *((uint4 *)(&scratch[loc])) = (c ? t : t2);
}

__device__ __forceinline__
void write_keys(const uint32_t b[4], const uint32_t bx[4], uint32_t *scratch, uint32_t start) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  start = scrypt_block*SCRYPT_SCRATCH_PER_BLOCK + (32*start) + 8*(threadIdx.x%4);
  write_keys_direct(b, bx, scratch, start);
}


__device__  __forceinline__ void read_keys_direct(uint32_t b[4], uint32_t bx[4], const uint32_t *scratch, uint32_t start) {

  uint4 t, t2;

  // Tricky bit: We do the work on behalf of thread+4, but then when
  // we steal, we have to steal from (thread+28)%32 to get the right
  // stuff back.
  start = __shfl((int)start, (threadIdx.x & 0x7c)) + 8*(threadIdx.x%4);

  int target_thread = (threadIdx.x + 4)%32;
  int t2_start = __shfl((int)start, target_thread) + 4;

  bool c = (threadIdx.x & 0x4);

  int loc = c ? t2_start : start;
  t = __ldg((uint4 *)(&scratch[loc]));
  loc = c ? start : t2_start;
  t2 = __ldg((uint4 *)(&scratch[loc]));

  uint4 tmp = t; t = (c ? t2 : t); t2 = (c ? tmp : t2);
  
  b[0] = t.x; b[1] = t.y; b[2] = t.z; b[3] = t.w;

  int steal_target = (threadIdx.x + 28)%32;

  bx[0] = __shfl((int)t2.x, steal_target);
  bx[1] = __shfl((int)t2.y, steal_target);
  bx[2] = __shfl((int)t2.z, steal_target);
  bx[3] = __shfl((int)t2.w, steal_target);
}


__device__  __forceinline__ void read_xor_keys_direct(uint32_t b[4], uint32_t bx[4], const uint32_t *scratch, uint32_t start) {

  uint4 t, t2;

  // Tricky bit: We do the work on behalf of thread+4, but then when
  // we steal, we have to steal from (thread+28)%32 to get the right
  // stuff back.
  start = __shfl((int)start, (threadIdx.x & 0x7c)) + 8*(threadIdx.x%4);

  int target_thread = (threadIdx.x + 4)%32;
  int t2_start = __shfl((int)start, target_thread) + 4;

  bool c = (threadIdx.x & 0x4);

  int loc = c ? t2_start : start;
  t = __ldg((uint4 *)(&scratch[loc]));
  loc = c ? start : t2_start;
  t2 = __ldg((uint4 *)(&scratch[loc]));

  uint4 tmp = t; t = (c ? t2 : t); t2 = (c ? tmp : t2);
  
  b[0] ^= t.x; b[1] ^= t.y; b[2] ^= t.z; b[3] ^= t.w;

  int steal_target = (threadIdx.x + 28)%32;

  bx[0] ^= __shfl((int)t2.x, steal_target);
  bx[1] ^= __shfl((int)t2.y, steal_target);
  bx[2] ^= __shfl((int)t2.z, steal_target);
  bx[3] ^= __shfl((int)t2.w, steal_target);
}


__device__  __forceinline__ void read_xor_keys(uint32_t b[4], uint32_t bx[4], const uint32_t *scratch, uint32_t start) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  start = scrypt_block*SCRYPT_SCRATCH_PER_BLOCK + (32*start);
  read_xor_keys_direct(b, bx, scratch, start);
}


__device__  __forceinline__ void primary_order_shuffle(uint32_t b[4], uint32_t bx[4]) {
  /* Inner loop shuffle targets */
  int x1_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+1)&0x3);
  int x2_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+2)&0x3);
  int x3_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+3)&0x3);
  
  b[3] = __shfl((int)b[3], x1_target_lane);
  b[2] = __shfl((int)b[2], x2_target_lane);
  b[1] = __shfl((int)b[1], x3_target_lane);
  uint32_t tmp = b[1]; b[1] = b[3]; b[3] = tmp;
  
  bx[3] = __shfl((int)bx[3], x1_target_lane);
  bx[2] = __shfl((int)bx[2], x2_target_lane);
  bx[1] = __shfl((int)bx[1], x3_target_lane);
  tmp = bx[1]; bx[1] = bx[3]; bx[3] = tmp;
}

/*
 * load_key loads a 32*32bit key from a contiguous region of memory in B.
 * The input keys are in external order (i.e., 0, 1, 2, 3, ...).
 * After loading, each thread has its four b and four bx keys stored
 * in internal processing order.
 */

__device__  __forceinline__ void load_key(const uint32_t *B, uint32_t b[4], uint32_t bx[4]) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int key_offset = scrypt_block * 32;
  uint32_t thread_in_block = threadIdx.x % 4;

  // Read in permuted order. Key loads are not our bottleneck right now.
#pragma unroll 4
  for (int i = 0; i < 4; i++) {
    b[i] = B[key_offset + 4*thread_in_block + (thread_in_block+i)%4];
    bx[i] = B[key_offset + 4*thread_in_block + (thread_in_block+i)%4 + 16];
  }

  primary_order_shuffle(b, bx);
  
}

/*
 * store_key performs the opposite transform as load_key, taking
 * internally-ordered b and bx and storing them into a contiguous
 * region of B in external order.
 */

__device__  __forceinline__ void store_key(uint32_t *B, uint32_t b[4], uint32_t bx[4]) {
  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int key_offset = scrypt_block * 32;
  uint32_t thread_in_block = threadIdx.x % 4;

  primary_order_shuffle(b, bx);

#pragma unroll 4
  for (int i = 0; i < 4; i++) {
    B[key_offset + 4*thread_in_block + (thread_in_block+i)%4] = b[i];
    B[key_offset + 4*thread_in_block + (thread_in_block+i)%4 + 16] = bx[i];
  }
}


/*
 * salsa_xor_core does the equivalent of the xor_salsa8 loop from
 * tarsnap's implementation of scrypt. The original scrypt called:
 * xor_salsa8(&X[0], &X[16]); <-- the "b" loop
 * xor_salsa8(&X[16], &X[0]); <-- the "bx" loop
 * This version is unrolled to handle both of these loops in a single
 * call to avoid unnecessary data movement.
 */

__device__  __forceinline__ void salsa_xor_core(uint32_t b[4], uint32_t bx[4],
                                 const int x1_target_lane,
                                 const int x2_target_lane,
                                 const int x3_target_lane) {
    uint32_t x[4];

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      b[i] ^= bx[i];
      x[i] = b[i];
    }

    // Enter in "column" mode (t0 has 0, 4, 8, 12)

#pragma unroll 4
    for (int j = 0; j < 4; j++) {
    
      // Mixing phase of salsa
      XOR_ROTATE_ADD(1, 0, 3, 7);
      XOR_ROTATE_ADD(2, 1, 0, 9);
      XOR_ROTATE_ADD(3, 2, 1, 13);
      XOR_ROTATE_ADD(0, 3, 2, 18);
      
      /* Transpose rows and columns. */
      /* Unclear if this optimization is needed: These are ordered based
       * upon the dependencies needed in the later xors. Compiler should be
       * able to figure this out, but might as well give it a hand. */
      x[1] = __shfl((int)x[1], x3_target_lane);
      x[3] = __shfl((int)x[3], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      
      /* The next XOR_ROTATE_ADDS could be written to be a copy-paste of the first,
       * but the register targets are rewritten here to swap x[1] and x[3] so that
       * they can be directly shuffled to and from our peer threads without
       * reassignment. The reverse shuffle then puts them back in the right place.
       */
      
      XOR_ROTATE_ADD(3, 0, 1, 7);
      XOR_ROTATE_ADD(2, 3, 0, 9);
      XOR_ROTATE_ADD(1, 2, 3, 13);
      XOR_ROTATE_ADD(0, 1, 2, 18);
      
      x[3] = __shfl((int)x[3], x3_target_lane);
      x[1] = __shfl((int)x[1], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
    }

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      b[i] += x[i];
      // The next two lines are the beginning of the BX-centric loop iteration
      bx[i] ^= b[i];
      x[i] = bx[i];
    }

    // This is a copy of the same loop above, identical but stripped of comments.
    // Duplicated so that we can complete a bx-based loop with fewer register moves.
#pragma unroll 4
    for (int j = 0; j < 4; j++) {
      XOR_ROTATE_ADD(1, 0, 3, 7);
      XOR_ROTATE_ADD(2, 1, 0, 9);
      XOR_ROTATE_ADD(3, 2, 1, 13);
      XOR_ROTATE_ADD(0, 3, 2, 18);
      
      x[1] = __shfl((int)x[1], x3_target_lane);
      x[3] = __shfl((int)x[3], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
      
      XOR_ROTATE_ADD(3, 0, 1, 7);
      XOR_ROTATE_ADD(2, 3, 0, 9);
      XOR_ROTATE_ADD(1, 2, 3, 13);
      XOR_ROTATE_ADD(0, 1, 2, 18);
      
      x[3] = __shfl((int)x[3], x3_target_lane);
      x[1] = __shfl((int)x[1], x1_target_lane);
      x[2] = __shfl((int)x[2], x2_target_lane);
    }

    // At the end of these iterations, the data is in primary order again.
#undef XOR_ROTATE_ADD

#pragma unroll 4
    for (int i = 0; i < 4; i++) {
      bx[i] += x[i];
    }
}


/*
 * The hasher_gen_kernel operates on a group of 1024-bit input keys
 * in B, stored as:
 * B = { k1B k1Bx k2B k2Bx ... }
 * and fills up the scratchpad with the iterative hashes derived from
 * those keys:
 * scratch { k1h1B k1h1Bx K1h2B K1h2Bx ... K2h1B K2h1Bx K2h2B K2h2Bx ... }
 * scratch is 1024 times larger than the input keys B.
 * It is extremely important to stream writes effectively into scratch;
 * less important to coalesce the reads from B.
 *
 * Key ordering note: Keys are input from B in "original" order:
 * K = {k1, k2, k3, k4, k5, ..., kx15, kx16, kx17, ..., kx31 }
 * After inputting into kernel_gen, each component k and kx of the
 * key is transmuted into a permuted internal order to make processing faster:
 * K = k, kx with:
 * k = 0, 4, 8, 12, 5, 9, 13, 1, 10, 14, 2, 6, 15, 3, 7, 11
 * and similarly for kx.
 */

__global__
void kepler_scrypt_core_kernelA(const uint32_t *d_idata, uint32_t *scratch) {

  /* Each thread operates on four of the sixteen B and Bx variables. Thus,
   * each key is processed by four threads in parallel. salsa_scrypt_core
   * internally shuffles the variables between threads (and back) as
   * needed.
   */
  uint32_t b[4], bx[4];

  load_key(d_idata, b, bx);
  
  /* Inner loop shuffle targets */
  int x1_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+1)&0x3);
  int x2_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+2)&0x3);
  int x3_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+3)&0x3);

  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int start = scrypt_block*SCRYPT_SCRATCH_PER_BLOCK + 8*(threadIdx.x%4);

  write_keys_direct(b, bx, scratch, start);
  for (int i = 1; i < 1024; i++) {
    salsa_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane);
    write_keys_direct(b, bx, scratch, start+32*i);
  }
}


/*
 * hasher_hash_kernel runs the second phase of scrypt after the scratch
 * buffer is filled with the iterative hashes: It bounces through
 * the scratch buffer in pseudorandom order, mixing the key as it goes.
 */

__global__
void kepler_scrypt_core_kernelB(uint32_t *d_odata, const uint32_t *scratch) {

  /* Each thread operates on a group of four variables that must be processed
   * together. Shuffle between threaads in a warp between iterations.
   */
  uint32_t b[4], bx[4];

  int scrypt_block = (blockIdx.x*blockDim.x + threadIdx.x)/THREADS_PER_SCRYPT_BLOCK;
  int start = scrypt_block*SCRYPT_SCRATCH_PER_BLOCK + 8*(threadIdx.x%4);

  read_keys_direct(b, bx, scratch, start+32*1023);

  /* Inner loop shuffle targets */
  int x1_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+1)&0x3);
  int x2_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+2)&0x3);
  int x3_target_lane = (threadIdx.x & 0xfc) + (((threadIdx.x & 0x03)+3)&0x3);

  salsa_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane);

  for (int i = 0; i < 1024; i++) {

    // Bounce through the key space and XOR the new keys in.
    // Critical thing: (X[16] & 1023) tells us the next slot to read.
    // X[16] in the original is bx[0]
    int slot = bx[0] & 1023;
    read_xor_keys(b, bx, scratch, slot);
    salsa_xor_core(b, bx, x1_target_lane, x2_target_lane, x3_target_lane);
  }

  store_key(d_odata, b, bx);
}

// scratchbuf constants (pointers to scratch buffer for each work unit)

KeplerKernel::KeplerKernel() : KernelInterface()
{
}

static uint32_t *d_scratch;

void KeplerKernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    // this currently REQUIRES single memory allocation mode (-m 1 flag)
    d_scratch = h_V[0];
}

bool KeplerKernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // this kernel needs 4 threads per work unit. We scale up the grid x dimension to compensate.
    grid.x *= 4;

    // First phase: Sequential writes to scratchpad.

    kepler_scrypt_core_kernelA<<< grid, threads, 0, stream >>>(d_idata, d_scratch);

    // Optional millisecond sleep in between kernels

    if (!benchmark && interactive) {
        checkCudaErrors(MyStreamSynchronize(stream, 1, thr_id));
#ifdef WIN32
        Sleep(1);
#else
        usleep(1000);
#endif
    }

    // Second phase: Random read access from scratchpad.

    kepler_scrypt_core_kernelB<<< grid, threads, 0, stream >>>(d_odata, d_scratch);

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}
